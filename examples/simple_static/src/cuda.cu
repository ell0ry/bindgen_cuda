#include "hip/hip_runtime.h"
#include "stdio.h"

extern "C" __global__ void sin_kernel(float *out, const float *inp, const int32_t numel) {
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < numel) {
        out[i] = sin(inp[i]);
    }
}

extern "C" int launch_sin2(float *out, const float *inp, int32_t n,  hipStream_t stream) {
    sin_kernel<<<n, 1, 0>>>(out, inp, n); 
    return 0;
}
